#include "cuda/memory.hpp"
#include "cuda/tensor_ops.hpp"
#include "cuda/tensor_ops_kernels.hpp"
#include "cuda/util.hpp"
#include "defs.hpp"
#include <array>
#include <hip/hip_runtime.h>

using namespace bnmf_algs;

template <typename T>
void cuda::tensor_sums(const cuda::DeviceMemory1D<T>& tensor,
                       const shape<3>& dims,
                       std::array<cuda::DeviceMemory1D<T>, 3>& result_arr) {
    // input tensor properties
    const long x = dims[0];
    const long y = dims[1];
    const long z = dims[2];

    // create GPU stream and device (default stream)
    Eigen::CudaStreamDevice stream;
    Eigen::GpuDevice dev(&stream);

    // sum axis
    shape<1> sum_axis;

    // map GPU tensor to Eigen (no copying)
    Eigen::TensorMap<tensor_t<T, 3>> in_tensor(tensor.data(), x, y, z);

    // compute sum along each axis
    for (size_t axis = 0; axis < 3; ++axis) {
        long rows = (axis == 0) ? y : x;
        long cols = (axis == 2) ? y : z;

        // map GPU tensor to Eigen (no copying)
        Eigen::TensorMap<tensor_t<T, 2>> out_tensor(result_arr[axis].data(),
                                                    rows, cols);

        // axis to sum along
        sum_axis[0] = axis;

        // sum the tensor on GPU
        out_tensor.device(dev) = in_tensor.sum(sum_axis);
        hipStreamSynchronize(stream.stream());
    }
}

template <typename Real> void cuda::apply_psi(DeviceMemory1D<Real>& range) {
    // grid/block dimensions
    const auto num_elems = range.dims()[0];
    size_t threads_per_block = 1024;
    size_t blocks_per_grid = cuda::idiv_ceil(num_elems, threads_per_block);

    // apply kernel
    kernel::apply_psi<<<blocks_per_grid, threads_per_block>>>(range.data(),
                                                              num_elems);
    auto err = hipGetLastError();
    BNMF_ASSERT(err == hipSuccess, "Error running kernel in cuda::apply_psi");
}

template <typename T>
void cuda::bld_mult::update_grad_plus(const tensor_t<T, 3>& S,
                                      const matrix_t<T>& beta_eph,
                                      tensor_t<T, 3>& grad_plus) {
    // tensor dimensions
    const auto x = static_cast<size_t>(S.dimension(0));
    const auto y = static_cast<size_t>(S.dimension(1));
    const auto z = static_cast<size_t>(S.dimension(2));

    // create host memory wrappers
    HostMemory3D<const T> host_S(S.data(), x, y, z);
    HostMemory3D<T> host_grad_plus(grad_plus.data(), x, y, z);
    HostMemory2D<const T> host_beta_eph(beta_eph.data(), y, z);

    // allocate device memory
    DeviceMemory3D<T> device_S(x, y, z);
    DeviceMemory2D<T> device_beta_eph(y, z);
    DeviceMemory3D<T> device_grad_plus(x, y, z);

    // copy S to GPU
    copy3D(device_S, host_S, hipMemcpyHostToDevice);

    // copy beta_eph to GPU
    copy2D(device_beta_eph, host_beta_eph, hipMemcpyHostToDevice);

    // block dimensions (number of threads per block axis)
    constexpr size_t block_size_x = 16;
    constexpr size_t block_size_y = 16;
    constexpr size_t block_size_z = 4;
    dim3 block_dims(block_size_y, block_size_x, block_size_z);
    dim3 grid_dims(cuda::idiv_ceil(y, block_size_y),
                   cuda::idiv_ceil(x, block_size_x),
                   cuda::idiv_ceil(z, block_size_z));

    // run kernel
    kernel::update_grad_plus<<<grid_dims, block_dims>>>(
        device_S.pitched_ptr(), device_beta_eph.data(), device_beta_eph.pitch(),
        device_grad_plus.pitched_ptr(), y, x, z);
    auto err = hipGetLastError();
    BNMF_ASSERT(err == hipSuccess,
                "Error running kernel in cuda::bld_mult::update_grad_plus");

    // copy result onto grad_plus
    copy3D(host_grad_plus, device_grad_plus, hipMemcpyDeviceToHost);
}

/************************ TEMPLATE INSTANTIATIONS *****************************/
template void
cuda::tensor_sums<double>(const cuda::DeviceMemory1D<double>&, const shape<3>&,
                          std::array<cuda::DeviceMemory1D<double>, 3>&);
template void
cuda::tensor_sums<float>(const cuda::DeviceMemory1D<float>&, const shape<3>&,
                         std::array<cuda::DeviceMemory1D<float>, 3>&);
template void cuda::tensor_sums<int>(const cuda::DeviceMemory1D<int>&,
                                     const shape<3>&,
                                     std::array<cuda::DeviceMemory1D<int>, 3>&);
template void
cuda::tensor_sums<long>(const cuda::DeviceMemory1D<long>&, const shape<3>&,
                        std::array<cuda::DeviceMemory1D<long>, 3>&);
template void
cuda::tensor_sums<size_t>(const cuda::DeviceMemory1D<size_t>&, const shape<3>&,
                          std::array<cuda::DeviceMemory1D<size_t>, 3>&);

template void cuda::apply_psi<double>(cuda::DeviceMemory1D<double>&);
template void cuda::apply_psi<float>(cuda::DeviceMemory1D<float>&);

template void cuda::bld_mult::update_grad_plus<double>(
    const tensor_t<double, 3>&, const matrix_t<double>&, tensor_t<double, 3>&);
template void cuda::bld_mult::update_grad_plus<float>(const tensor_t<float, 3>&,
                                                      const matrix_t<float>&,
                                                      tensor_t<float, 3>&);
