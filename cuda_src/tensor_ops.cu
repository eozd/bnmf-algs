#include "cuda/memory.hpp"
#include "cuda/tensor_ops.hpp"
#include "cuda/tensor_ops_kernels.hpp"
#include "cuda/util.hpp"
#include <array>
#include <hip/hip_runtime.h>

using namespace bnmf_algs;

template <typename T>
void cuda::tensor_sums(const cuda::DeviceMemory3D<T>& tensor,
                       std::array<cuda::DeviceMemory2D<T>, 3>& result_arr) {
    // input tensor properties
    const auto dims = tensor.dims();
    const size_t height = dims[0];
    const size_t width = dims[1];
    const size_t depth = dims[2];

    // block dimensions (number of threads per block axis)
    constexpr size_t block_size_height = 32;
    constexpr size_t block_size_width = 32;
    constexpr size_t block_size_depth = 1;
    dim3 block_dims(block_size_width, block_size_height, block_size_depth);

    constexpr size_t num_axes = 3;
    // launch each sum kernel on different CUDA stream
    std::array<hipStream_t, num_axes> stream_arr;
    for (size_t i = 0; i < num_axes; ++i) {
        hipStreamCreate(&stream_arr[i]);
    }

    // compute sum along each axis
    for (size_t axis = 0; axis < num_axes; ++axis) {
        // rows, cols, layers set when looked at the tensor from the face along
        // the current axis
        const size_t n_rows = (axis == 0) ? width : height;
        const size_t n_cols = (axis == 2) ? width : depth;
        const size_t n_layers = dims[axis];

        // kernel grid dimensions
        dim3 grid_dims(cuda::idiv_ceil(n_cols, block_size_width),
                       cuda::idiv_ceil(n_rows, block_size_height), 1);

        // launch asynchronous kernel
        kernel::sum_tensor3D<<<grid_dims, block_dims, 0, stream_arr[axis]>>>(
            tensor.pitched_ptr(), result_arr[axis].data(),
            result_arr[axis].pitch(), axis, n_rows, n_cols, n_layers);
    }

    // synchronize all streams
    for (size_t i = 0; i < num_axes; ++i) {
        hipStreamSynchronize(stream_arr[i]);
        hipStreamDestroy(stream_arr[i]);
    }
}
/************************ TEMPLATE INSTANTIATIONS *****************************/
// We need these because nvcc requires explicit instantiations of all template
// functions.

// tensor_sums
template void cuda::tensor_sums(const cuda::DeviceMemory3D<double>&,
                                std::array<cuda::DeviceMemory2D<double>, 3>&);
template void cuda::tensor_sums(const cuda::DeviceMemory3D<float>&,
                                std::array<cuda::DeviceMemory2D<float>, 3>&);
template void cuda::tensor_sums(const cuda::DeviceMemory3D<int>&,
                                std::array<cuda::DeviceMemory2D<int>, 3>&);
template void cuda::tensor_sums(const cuda::DeviceMemory3D<long>&,
                                std::array<cuda::DeviceMemory2D<long>, 3>&);
template void cuda::tensor_sums(const cuda::DeviceMemory3D<size_t>&,
                                std::array<cuda::DeviceMemory2D<size_t>, 3>&);

